#include "hip/hip_runtime.h"
#include "UpdatePositions.h"

float * g_outputs_d, *g_sweepers_d_2;

void set_up_update_positions(int num_sweepers)
{
	hipMalloc((void **)&g_outputs_d, num_sweepers * 2 * sizeof(float));
	hipMalloc((void **)&g_sweepers_d_2, num_sweepers * 2 * sizeof(float));
}

void end_update_positions()
{
	hipFree(g_outputs_d);
	hipFree(g_sweepers_d_2);
}

void call_cuda_update_positions(int num_sweepers, float max_speed, float * outputs, float * sweepers)
{
	hipMemcpy(g_outputs_d, outputs, num_sweepers * 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(g_sweepers_d_2, sweepers, num_sweepers * 2 * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(2, 1, 1);
	dim3 blocks(num_sweepers, 1, 1);
	update_positions <<<blocks, threads >>>(max_speed, g_outputs_d, g_sweepers_d_2);

	hipMemcpy(sweepers, g_sweepers_d_2, num_sweepers * 2 * sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void update_positions(float max_speed, float * outputs_d, float * sweepers_d)
{
	int my_index = blockIdx.x * blockDim.x + threadIdx.x;

	sweepers_d[my_index] +=  (2 * outputs_d[my_index] * max_speed) - max_speed;
}